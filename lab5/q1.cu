#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void vectorAdd256(int *a, int *b, int *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int idx = blockIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N;
    printf("Enter the size of the vector: ");
    scanf("%d", &N);
    
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                     

    for (int i = 0; i < N; i++) {
        a[i] = i*i;
        b[i] = i;
    }

    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    vectorAdd<<<N, 1>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();  
    
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Results using N blocks: ");
    for (int i = 0; i < N && i < 10; i++) {  
        printf("%d ", c[i]);
    }
    printf("\n");

    vectorAdd256<<<1, N>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Results using N threads (1 block): ");
    for (int i = 0; i < N && i < 10; i++) { 
        printf("%d ", c[i]);
    }
    printf("\n");

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;  
    vectorAdd256<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize(); 

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Results using 256 threads per block: ");
    for (int i = 0; i < N && i < 10; i++) { 
        printf("%d ", c[i]);
    }
    printf("\n");

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
