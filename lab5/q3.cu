#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void computeSine(float* input, float* output, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < size) {
        output[idx] = sinf(input[idx]);  
    }
}

int main() {
    float h_input[5];  
    printf("Enter 5 angles in radians:\n"); 
    for (int i = 0; i < 5; i++) {
        printf("Angle %d: ", i + 1);
        scanf("%f", &h_input[i]);
    }

    int N = 5;  
    
    float* h_output = (float*)malloc(N * sizeof(float));  

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256); 
    dim3 numBlocks((N + blockSize.x - 1) / blockSize.x);  

    computeSine<<<numBlocks, blockSize>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("\nResults:\n");
    for (int i = 0; i < N; i++) {
        printf("Input angle (radians): %f -> sin: %f\n", h_input[i], h_output[i]);
    }

    hipFree(d_input);
    hipFree(d_output);

    free(h_output);

    return 0;
}
