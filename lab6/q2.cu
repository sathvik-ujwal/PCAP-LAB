#include<stdio.h>
#include<hip/hip_runtime.h>


#define N 10

__global__ void selectionSort(int * arr, int n) {
    for (int i = 0; i < n-1; i++) {
        int k = i;
        for (int j = i+1;  j < n; j++) {
            if (arr[j] < arr[k]) {
                k = j;
            }
        }

        if (k != i) {
            int temp = arr[i];
            arr[i] = arr[k];
            arr[k] = temp;
        }
    }
}

int main() {
    int input_arr[N] = {64,  25, 12, 22, 11, 90, 55, 33, 77, 88};
    int *d_arr;

    hipMalloc((void**)&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, input_arr, N*sizeof(int), hipMemcpyHostToDevice);

    selectionSort<<<1,1>>>(d_arr, N);

    hipMemcpy(input_arr, d_arr, N* sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array ");
    for (int i = 0; i < N; i++) {
        printf("%d ", input_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}