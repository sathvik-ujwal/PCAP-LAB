#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void convolution_1D_basic_kernel(float *N, float *M, float *P, int Mask_Width, int Width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    float Pvalue = 0;
    int N_start_point = i - (Mask_Width/2);
    for (int j = 0; j < Mask_Width; j++ ) {
        if (N_start_point + j >= 0 && N_start_point + j < Width) {
            Pvalue += N[N_start_point + j] * M[j];
        }
    }
    P[i] = Pvalue;
}

int main() {
    int N, M;
    printf("Enter the size of the array\n");
    scanf("%d", &N);
    printf("Enter the size of the convolution mask\n");
    scanf("%d", &M);

    float inputArr[N];
    float Mask[M];
    float resultantArr[N];

    printf("Enter %d elements for input array\n", N);
    for (int i = 0 ; i < N ; i++) {
        scanf("%f", &inputArr[i]);
    }
    printf("Enter %d elements of the Mask\n", M);
    for (int i = 0; i < M; i++) {
        scanf("%f", &Mask[i]);
    } 

    float *d_inputArr, *d_Mask, *d_resultArr;
    hipMalloc((void**)&d_inputArr, N * sizeof(float));
    hipMalloc((void**)&d_Mask, M * sizeof(float));
    hipMalloc((void**)&d_resultArr, N * sizeof(float));

    hipMemcpy(d_inputArr, inputArr, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Mask, Mask, M * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    convolution_1D_basic_kernel<<<gridSize, blockSize>>>(d_inputArr, d_Mask, d_resultArr, M, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR : %s \n", hipGetErrorString(err));
    }

    hipMemcpy(resultantArr, d_resultArr, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Convolution  result\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", resultantArr[i]);
    }
    printf("\n");

    hipFree(d_inputArr);
    hipFree(d_Mask);
    hipFree(d_resultArr);

    return 0;
}

