#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10 

__global__ void oddKernel(int *arr, int n) {
    int i = 2 * threadIdx.x + 1;
    if (i < n - 1 && arr[i] > arr[i + 1]) {
        int temp = arr[i];
        arr[i] = arr[i + 1];
        arr[i + 1] = temp;
    }
}

__global__ void evenKernel(int *arr, int n) {
    int i = 2 * threadIdx.x;
    if (i < n - 1 && arr[i] > arr[i + 1]) {
        int temp = arr[i];
        arr[i] = arr[i + 1];
        arr[i + 1] = temp;
    }
}

int main() {
    int h_arr[N] = {64, 25, 12, 22, 1, 90, 55, 564, 77, 88};
    int *d_arr;

    hipMalloc((void **)&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = N / 2;
    for (int i = 0; i < N; i++) {
        oddKernel<<<1, threadsPerBlock>>>(d_arr, N);
        hipDeviceSynchronize();
        evenKernel<<<1, threadsPerBlock>>>(d_arr, N);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}