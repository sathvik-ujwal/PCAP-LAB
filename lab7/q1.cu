#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_WORD_LEN 32
#define MAX_SENTENCE_LEN 256

__device__ bool str_cmp(const char *str1, const char *str2, int wordLen) {
    for (int i = 0; i < wordLen; i++) {
        if (str1[i] != str2[i]) {
            return false;
        }
    }
    return (str1[wordLen] == ' ' || str1[wordLen] == '\0' || str1[wordLen] == '.' || str1[wordLen] == ',');
}

__global__ void countWordOccurrences(char *sentence, char *word, int *count, int sentenceLen, int wordLen) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx <= sentenceLen - wordLen) {
        if (str_cmp(&sentence[idx], word, wordLen)) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    char h_sentence[MAX_SENTENCE_LEN];
    char h_word[MAX_WORD_LEN];
    int h_count = 0;

    printf("Enter a sentence: ");
    fgets(h_sentence, MAX_SENTENCE_LEN, stdin);
    h_sentence[strcspn(h_sentence, "\n")] = 0;

    printf("Enter the word to count: ");
    scanf("%s", h_word);

    char *d_sentence, *d_word;
    int *d_count;
    int sentenceLen = strlen(h_sentence);
    int wordLen = strlen(h_word);

    hipMalloc((void **)&d_sentence, sentenceLen + 1);
    hipMalloc((void **)&d_word, wordLen + 1);
    hipMalloc((void **)&d_count, sizeof(int));

    hipMemcpy(d_sentence, h_sentence, sentenceLen + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_word, h_word, wordLen + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (sentenceLen + blockSize - 1) / blockSize;
    countWordOccurrences<<<gridSize, blockSize>>>(d_sentence, d_word, d_count, sentenceLen, wordLen);
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", h_word, h_count);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}
