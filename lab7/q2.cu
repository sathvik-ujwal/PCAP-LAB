#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

__global__ void transformString(char *d_S, char *d_RS, int N) {
    int idx = threadIdx.x;
    int start_index = 0;

    for (int i = 0; i < N; i++) {
        if (idx < (N - i)) {
            d_RS[start_index + idx] = d_S[idx];
        }
        start_index += (N - i);
    }
}

int main() {
    char *h_S; 
    char *h_RS;
    int N, M;

    printf("Enter a string: ");
    h_S = (char *)malloc(1000 * sizeof(char)); 
    scanf("%s", h_S); 

    N = strlen(h_S);  
    M = (N * (N + 1)) / 2; 

    h_RS = (char *)malloc((M + 1) * sizeof(char)); 
    char *d_S, *d_RS;
    hipMalloc((void **)&d_S, N * sizeof(char));
    hipMalloc((void **)&d_RS, M * sizeof(char));

    hipMemcpy(d_S, h_S, N * sizeof(char), hipMemcpyHostToDevice);
    transformString<<<1, N>>>(d_S, d_RS, N);
    hipMemcpy(h_RS, d_RS, M * sizeof(char), hipMemcpyDeviceToHost);
    h_RS[M] = '\0'; 

    printf("Input string S: %s\n", h_S);
    printf("Output string RS: %s\n", h_RS);

    free(h_S);
    free(h_RS);
    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}