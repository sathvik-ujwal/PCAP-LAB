#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<conio.h>

#define N 1024

__global__ void CUDACount(char * A, unsigned int *d_count) {
    int i = threadIdx.x;
    if (A[i] == 'a') {
        atomicAdd(d_count, 1);
    }
}

int main() {
    char A[N];
    char *d_A;
    unsigned int * count = 0, *d_count, *result;
    printf("Enter a string\n");
    fgets(A, N, stdin);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, count, sizeof(unsigned int), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error1: %s\n", hipGetErrorString(error));
    }

    CUDACount<<<1, strlen(A)>>>(d_A, d_count);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error1: %s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elaspedTime;
    hipEventElapsedTime(&elaspedTime, start, stop);
    result = (unsigned int *)malloc(sizeof(unsigned int));
    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurances of a = %u\n", *result);
    printf("Time taken = %f\n", elaspedTime);
    hipFree(d_A);
    hipFree(d_count);
    printf("\n");
    free(result);
    return 0;

}